#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <fstream>
#include <iostream>
using namespace std;
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>


typedef struct {
  int idx;
  double *el;
} Matrix;

int main(int argc, char*argv[])
{

        if (argc<2) {
                cout << "Input file not specified. Please, specify it as a first argument." << endl;
                cout << "example: " << argv[0] << " train_data_matrix_T.txt" << endl;
                return -1;
        }
        ifstream file(argv[1]);
        ofstream S_diag("S_diag.txt");
        ofstream U_rows("U_rows.txt");
	ofstream VT_cols("VT_cols.txt");

        if (!file)
        {
                cout << "Error opening file" << endl;
                return -1;
        }

        int idx;
        file >> idx;                     
        if (argc>2) cout << "N=" << idx << endl;

	// --- gesvd only supports Nrows >= Ncols
	// --- column major memory ordering

        const int m = 1682; 
        const int n = 943; 
        const int lda = m;

        // --- CUDA solver initialization

        hipsolverHandle_t solver_handle; 
        hipblasHandle_t cublasH = NULL; 
   
        Matrix A, U, VT;      //host matrices
        A.el = new double[lda*n];    //.... 
        U.el = new double[lda*n];    //...
        VT.el = new double[lda*n];   //..

    //reading from file into matrices
        for (long i=0; i<(lda*n); i++){
                file >> A.el[i];
		double io = A.el[i];
                printf("%1.9f,  ",io);
	}
    printf("\n");
    
    double S[n]; // singular value
    double *d_rwork = NULL;

// --- cuSOLVE input/output parameters/arrays

    int work_size = 0;
    int info_gpu = 0;
    int *devInfo; hipMalloc ((void**)&devInfo, sizeof(int));

 
//  create cusolverDn/cublas handle
    assert(hipsolverDnCreate(&solver_handle));
    assert(hipblasCreate(&cublasH));


    // --- Setting the device matrix and moving the host matrix to the device

    double *d_A; hipMalloc ((void**)&d_A  , sizeof(double)*lda*n);
    hipMemcpy(d_A, A.el, sizeof(double)*lda*n, hipMemcpyHostToDevice);

    // --- device side SVD workspace and matrices

    double *d_S; hipMalloc ((void**)&d_S  , sizeof(double)*n);
    double *d_U; hipMalloc ((void**)&d_U  , sizeof(double)*lda*m);
    double *d_VT; hipMalloc ((void**)&d_VT , sizeof(double)*lda*n);
    
    double *d_W; hipMalloc ((void**)&d_W  , sizeof(double)*lda*n);


    // --- CUDA SVD initialization

    assert(hipsolverDnDgesvd_bufferSize(solver_handle,m,n,&work_size));
    double *d_work; assert(hipMalloc((void**)&d_work , sizeof(double)*work_size));


    // --- CUDA SVD execution

    assert(hipsolverDnDgesvd (solver_handle,'A','A',m,n,d_A,lda,d_S,d_U,lda,d_VT,lda,d_work,work_size,d_rwork,devInfo));
    assert(hipDeviceSynchronize());


    // --- Moving the results from device to host

    assert(hipMemcpy(U.el , d_U , sizeof(double)*lda*m, hipMemcpyDeviceToHost));
    assert(hipMemcpy(VT.el, d_VT, sizeof(double)*lda*n, hipMemcpyDeviceToHost));
    assert(hipMemcpy(S , d_S , sizeof(double)*n, hipMemcpyDeviceToHost));
    assert(hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost));


    assert(0 == info_gpu);
    printf("Singular values\n");
    for (int k=0; k<n; k++){
                        S_diag << S[k] << " ";
    }
    
    printf("=====\n");
    printf("\nLeft singular vectors - For y = A * x, the columns of U span the space of y\n");
    for (int k=0; k<m; k++){
                for (int j=0; j<m; j++)
                        U_rows << U.el[k+j*lda] << " ";
                U_rows << endl;
    }
    
    printf("=====\n");
    printf("\nRight singular vectors - For y = A * x, the columns of V span the space of x\n");
    for (int k=0; k<n; k++){
                for (int j=0; j<n; j++)
                        VT_cols << VT.el[k+j*lda] << " ";
                VT_cols << endl;
    }


    if (solver_handle) hipsolverDnDestroy(solver_handle);

    hipDeviceReset();

    return 0;
}






