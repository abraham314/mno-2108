#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<iostream>
#include<iomanip>
#include<stdlib.h>
#include<stdio.h>
#include<assert.h>
#include<math.h>

#include <hipsolver.h>
#include <hip/hip_runtime_api.h>

#include "Utilities.cuh"

/********/
/* MAIN */
/********/
int main(){

    // --- gesvd only supports Nrows >= Ncols
    // --- column major memory ordering

    const int Nrows = 7;
    const int Ncols = 5;

    // --- cuSOLVE input/output parameters/arrays
    int work_size = 0;
    int *devInfo;           gpuErrchk(hipMalloc(&devInfo,          sizeof(int)));

    // --- CUDA solver initialization
    hipsolverHandle_t solver_handle;
    hipsolverDnCreate(&solver_handle);

    // --- Setting the host, Nrows x Ncols matrix
    double *h_A = (double *)malloc(Nrows * Ncols * sizeof(double));
    for(int j = 0; j < Nrows; j++)
        for(int i = 0; i < Ncols; i++)
            h_A[j + i*Nrows] = (i + j*j) * sqrt((double)(i + j));

    // --- Setting the device matrix and moving the host matrix to the device
    double *d_A;            gpuErrchk(hipMalloc(&d_A,      Nrows * Ncols * sizeof(double)));
    gpuErrchk(hipMemcpy(d_A, h_A, Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice));

    // --- host side SVD results space
    double *h_U = (double *)malloc(Nrows * Nrows     * sizeof(double));
    double *h_V = (double *)malloc(Ncols * Ncols     * sizeof(double));
    double *h_S = (double *)malloc(min(Nrows, Ncols) * sizeof(double));

    // --- device side SVD workspace and matrices
    double *d_U;            gpuErrchk(hipMalloc(&d_U,  Nrows * Nrows     * sizeof(double)));
    double *d_V;            gpuErrchk(hipMalloc(&d_V,  Ncols * Ncols     * sizeof(double)));
    double *d_S;            gpuErrchk(hipMalloc(&d_S,  min(Nrows, Ncols) * sizeof(double)));

    // --- CUDA SVD initialization
    cusolveSafeCall(hipsolverDnDgesvd_bufferSize(solver_handle, Nrows, Ncols, &work_size));
    double *work;   gpuErrchk(hipMalloc(&work, work_size * sizeof(double)));

    // --- CUDA SVD execution
    cusolveSafeCall(hipsolverDnDgesvd(solver_handle, 'A', 'A', Nrows, Ncols, d_A, Nrows, d_S, d_U, Nrows, d_V, Ncols, work, work_size, NULL, devInfo));
    int devInfo_h = 0;  gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (devInfo_h != 0) std::cout   << "Unsuccessful SVD execution\n\n";

    // --- Moving the results from device to host
    gpuErrchk(hipMemcpy(h_S, d_S, min(Nrows, Ncols) * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_U, d_U, Nrows * Nrows     * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_V, d_V, Ncols * Ncols     * sizeof(double), hipMemcpyDeviceToHost));

    std::cout << "Singular values\n";
    for(int i = 0; i < min(Nrows, Ncols); i++) 
        std::cout << "d_S["<<i<<"] = " << std::setprecision(15) << h_S[i] << std::endl;

    std::cout << "\nLeft singular vectors - For y = A * x, the columns of U span the space of y\n";
    for(int j = 0; j < Nrows; j++) {
        printf("\n");
        for(int i = 0; i < Nrows; i++)
            printf("U[%i,%i]=%f\n",i,j,h_U[j*Nrows + i]);
    }

    std::cout << "\nRight singular vectors - For y = A * x, the columns of V span the space of x\n";
    for(int i = 0; i < Ncols; i++) {
        printf("\n");
        for(int j = 0; j < Ncols; j++)
            printf("V[%i,%i]=%f\n",i,j,h_V[j*Ncols + i]);
    }

    hipsolverDnDestroy(solver_handle);

    return 0;

}
