#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipsolver.h"

#define TEST_PASSED  0
#define TEST_FAILED  1

int main (void){
    /* matrix A */
    int n;
    int nnzA;
    int *Ap=NULL;
    int *Ai=NULL;
    double *Ax=NULL;
    int *d_Ap=NULL;
    int *d_Ai=NULL;
    double *d_rAx=NULL;
    /* matrices L and U */
    int nnzL, nnzU;
    int *Lp=NULL;
    int *Li=NULL;
    double* Lx=NULL;
    int *Up=NULL;
    int *Ui=NULL;
    double* Ux=NULL;
    /* reordering matrices */
    int *P=NULL;
    int *Q=NULL;
    int * d_P=NULL;
    int * d_Q=NULL;
    /* solution and rhs */
    int nrhs; //# of rhs for each system (currently only =1 is supported)
    double *d_X=NULL;
    double *d_T=NULL;
    /* cuda */
    hipError_t cudaStatus;
    /* cuolverRf */
    hipsolverRfHandle_t gH=NULL;
    hipsolverStatus_t status;
    /* host sparse direct solver */
    /* ... */
    /* other variables */
    int tnnzL, tnnzU;
    int *tLp=NULL;
    int *tLi=NULL;
    double *tLx=NULL;
    int *tUp=NULL;
    int *tUi=NULL;
    double *tUx=NULL;
    clock_t t1, t2;



    /* ASSUMPTION: recall that we are solving a set of linear systems
       A_{i} x_{i} = f_{i}  for i=0,...,k-1
       where the sparsity pattern of the coefficient matrices A_{i}
       as well as the reordering to minimize fill-in and the pivoting
       used during the LU factorization remain the same. */


    /* Step 1: solve the first linear system (i=0) on the host,
               using host sparse direct solver, which involves
               full LU factorization and solve. */
    /* ... */


    /* Step 2: interface to the library by extracting the following
               information from the first solve:
               a) triangular factors L and U
               b) pivoting and reordering permutations P and Q
               c) also, allocate all the necessary memory */
    /* ... */


    /* Step 3: use the library to solve subsequent (i=1,...,k-1) linear systems
    a) the  library setup (called only once) */
    //create handle
    status = hipsolverRfCreate(&gH);
    if (status != HIPSOLVER_STATUS_SUCCESS){
        printf ("[cusolverRf status %d]\n",status);
        return TEST_FAILED;
    }

    //set fast mode
    status = hipsolverRfSetResetValuesFastMode(gH,HIPSOLVERRF_RESET_VALUES_FAST_MODE_ON);
    if (status != HIPSOLVER_STATUS_SUCCESS){
        printf ("[cusolverRf status %d]\n",status);
        return TEST_FAILED;
    }


    //assemble internal data structures (you should use the coeffcient matrix A
    //corresponding to the second (i=1) linear system in this call)
    t1 = clock();
    status = hipsolverRfSetupHost(n, nnzA, Ap, Ai, Ax,
                               nnzL, Lp, Li, Lx, nnzU, Up, Ui, Ux, P, Q, gH);
    cudaStatus = hipDeviceSynchronize();
    t2 = clock();
    if ((status != HIPSOLVER_STATUS_SUCCESS) || (cudaStatus != hipSuccess)) {
        printf ("[cusolverRf status %d]\n",status);
        return TEST_FAILED;
    }
    printf("hipsolverRfSetupHost time = %f (s)\n", (t2-t1)/(float)CLOCKS_PER_SEC);

    //analyze available parallelism
    t1 = clock();
    status = hipsolverRfAnalyze(gH);
    cudaStatus = hipDeviceSynchronize();
    t2 = clock();
    if ((status != HIPSOLVER_STATUS_SUCCESS) || (cudaStatus != hipSuccess)) {
        printf ("[cusolverRf status %d]\n",status);
        return TEST_FAILED;
    }
    printf("hipsolverRfAnalyze time = %f (s)\n", (t2-t1)/(float)CLOCKS_PER_SEC);

    /* b) The  library subsequent (i=1,...,k-1) LU re-factorization
          and solve (called multiple times). */
    int k = 2;
    for (int i=1; i<k; i++){
        //LU re-factorization
        t1 = clock();
        status = hipsolverRfRefactor(gH);
        cudaStatus = hipDeviceSynchronize();
        t2 = clock();
        if ((status != HIPSOLVER_STATUS_SUCCESS) || (cudaStatus != hipSuccess)) {
            printf ("[cusolverRF status %d]\n",status);
            return TEST_FAILED;
        }
        printf("cuSolverReRefactor time = %f (s)\n", (t2-t1)/(float)CLOCKS_PER_SEC);

        //forward and backward solve
        t1 = clock();
        status = hipsolverRfSolve(gH, d_P, d_Q, nrhs, d_T, n, d_X, n);
        cudaStatus = hipDeviceSynchronize();
        t2 = clock();
        if ((status != HIPSOLVER_STATUS_SUCCESS) || (cudaStatus != hipSuccess)) {
            printf ("[cusolverRf status %d]\n",status);
            return TEST_FAILED;
        }
        printf("hipsolverRfSolve time = %f (s)\n", (t2-t1)/(float)CLOCKS_PER_SEC);

        // extract the factors (if needed)
        status = hipsolverRfExtractSplitFactorsHost(gH, &tnnzL, &tLp, &tLi, &tLx,
                                                &tnnzU, &tUp, &tUi, &tUx);
        if(status != HIPSOLVER_STATUS_SUCCESS){
            printf ("[cusolverRf status %d]\n",status);
            return TEST_FAILED;
        }
        /*
        //print
        int row, j;
        printf("printing L\n");
        for (row=0; row<n; row++){
            for (j=tLp[row]; j<tLp[row+1]; j++){
                printf("\%d,\%d,\%f\n",row,tLi[j],tLx[j]);
            }
        }
        printf("printing U\n");
        for (row=0; row<n; row++){
            for (j=tUp[row]; j<tUp[row+1]; j++){
                printf("\%d,\%d,\%f\n",row,tUi[j],tUx[j]);
            }
        }
        */

        /* perform any other operations based on the solution */
        /* ... */

        /* check if done */
        /* ... */

        /* proceed to solve the next linear system */
        // update the coefficient matrix using reset values
        // (assuming that the new linear system, in other words,
        //  new values are already on the GPU in the array d_rAx)
        t1 = clock();
        status = hipsolverRfResetValues(n,nnzA,d_Ap,d_Ai,d_rAx,d_P,d_Q,gH);
        cudaStatus = hipDeviceSynchronize();
        t2 = clock();
        if ((status != HIPSOLVER_STATUS_SUCCESS) || (cudaStatus != hipSuccess)) {
            printf ("[cusolverRf status %d]\n",status);
            return TEST_FAILED;
        }
        printf("cusolverRf_reset_values time = %f (s)\n", (t2-t1)/(float)CLOCKS_PER_SEC);
    }

    /* free memory and exit */
    /* ... */
    return TEST_PASSED;
}

